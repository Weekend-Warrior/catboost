#include "hip/hip_runtime.h"
#include "add_model_value.cuh"

#include <catboost/cuda/cuda_util/kernel/instructions.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <catboost/cuda/gpu_data/gpu_structures.h>


namespace NKernel {

    //it may be faster to cache in shared memory binValues, but for fold + bin + depth > 10 it'll be slower and may fail on shared memory allocation
    //so current impl more conservative with const-load cache
    template <int BLOCK_SIZE, int ELEMENTS_PER_THREAD>
    __launch_bounds__(BLOCK_SIZE, 2048 / BLOCK_SIZE)
    __global__ void AddBinModelValueImpl(const float* binValues, ui32 binCount,
                                         const ui32* bins, ui32 size,
                                         const ui32* readIndices, const ui32* writeIndices,
                                         float* cursor) {
        const ui32 i = blockIdx.x * BLOCK_SIZE * ELEMENTS_PER_THREAD + threadIdx.x;

        ui32 writeIndicesLocal[ELEMENTS_PER_THREAD];
        ui32 binsLocal[ELEMENTS_PER_THREAD];

        #pragma unroll ELEMENTS_PER_THREAD
        for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
            const int idx = i + j * BLOCK_SIZE;
            const ui32 readIdx = idx < size ? (readIndices ? LdgWithFallback(readIndices, idx) : idx) : (ui32)-1;;
            writeIndicesLocal[j] = idx < size ? (writeIndices ? LdgWithFallback(writeIndices, idx) : idx) : (ui32)-1;
            binsLocal[j] = idx < size ? LdgWithFallback(bins, readIdx) : 0;
        }

        float binsValuesLocal[ELEMENTS_PER_THREAD];

        #pragma unroll ELEMENTS_PER_THREAD
        for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
            const int idx = i + j * BLOCK_SIZE;
            binsValuesLocal[j] = idx < size ? LdgWithFallback(binValues, binsLocal[j])  : 0;
        }

        #pragma unroll ELEMENTS_PER_THREAD
        for (int j = 0; j < ELEMENTS_PER_THREAD; ++j) {
            const int idx = i + j * BLOCK_SIZE;
            if (idx < size) {
                cursor[writeIndicesLocal[j]] += binsValuesLocal[j];
            }
        }
    }

    void AddBinModelValue(const float* binValues, ui32 binCount,
                          const ui32* bins,
                          const ui32* readIndices, const ui32* writeIndices,
                          float* cursor, ui32 size,
                          TCudaStream stream) {
        const ui32 blockSize = 256;
        const ui32 elementsPerThreads = 10;
        const ui32 numBlocks = CeilDivide<ui32>(size, blockSize * elementsPerThreads);
        AddBinModelValueImpl<blockSize, elementsPerThreads> << <numBlocks, blockSize, 0, stream>>>(binValues, binCount, bins, size, readIndices, writeIndices, cursor);
    }



    __global__ void AddObliviousTreeImpl(const TCFeature* features, const ui8* bins, const float* leaves, ui32 depth,
                                         const ui32* cindex,
                                         const ui32* readIndices,
                                         const ui32* writeIndices,
                                         float* cursor,
                                         ui32 size) {

        ui32 tid = blockDim.x * blockIdx.x + threadIdx.x;

        __shared__ ui32 masksLocal[32];
        __shared__ ui32 valuesLocal[32];
        __shared__ ui64 offsetsLocal[32];
        __shared__ ui32 takeEqual[32];

        if (threadIdx.x < depth) {
            const int level = threadIdx.x;
            TCFeature feature = features[level];
            const ui32 value =(ui32)(bins[level]) << feature.Shift;
            const ui32 mask = feature.Mask << feature.Shift;

            masksLocal[level] = mask;
            valuesLocal[level] = value;
            takeEqual[level] = feature.OneHotFeature;
            offsetsLocal[level] = feature.Offset;
        }
        __syncthreads();

        while (tid < size) {
            ui32 bin = 0;
            const ui32 loadIdx = readIndices ? readIndices[tid] : tid;

            #pragma unroll 8
            for (ui32 level = 0; level < depth; ++level) {
                const ui32 value = valuesLocal[level];
                const ui32 mask = masksLocal[level];
                const ui32 featureVal = __ldg((cindex + offsetsLocal[level]) + loadIdx) & mask;
                const ui32 split = (takeEqual[level] ? (featureVal == value) : featureVal > value);
                bin |= split << level;
            }
            const ui32 writeIdx = writeIndices ? writeIndices[tid] : tid;
            cursor[writeIdx] += __ldg(leaves + bin);
            tid += blockDim.x  * gridDim.x;
        }
    }


    __global__ void ComputeObliviousTreeBinsImpl(const TCFeature* features, const ui8* bins,  ui32 depth,
                                                 const ui32* cindex,
                                                 const ui32* readIndices,
                                                 const ui32* writeIndices,
                                                 ui32* cursor,
                                                 ui32 size) {

        ui32 tid = blockDim.x * blockIdx.x + threadIdx.x;

        __shared__ ui32 masksLocal[32];
        __shared__ ui32 valuesLocal[32];
        __shared__ ui64 offsetsLocal[32];
        __shared__ ui32 takeEqual[32];

        if (threadIdx.x < depth) {
            const int level = threadIdx.x;
            TCFeature feature = features[level];
            const ui32 value =(ui32)(bins[level]) << feature.Shift;
            const ui32 mask = feature.Mask << feature.Shift;

            masksLocal[level] = mask;
            valuesLocal[level] = value;
            takeEqual[level] = feature.OneHotFeature;
            offsetsLocal[level] = feature.Offset;
        }
        __syncthreads();

        while (tid < size) {
            ui32 bin = 0;
            const ui32 loadIdx = readIndices ? readIndices[tid] : tid;

            #pragma unroll 8
            for (ui32 level = 0; level < depth; ++level) {
                const ui32 value = valuesLocal[level];
                const ui32 mask = masksLocal[level];
                const ui32 featureVal = __ldg(cindex + offsetsLocal[level] + loadIdx) & mask;
                const ui32 split = (takeEqual[level] ? (featureVal == value) : featureVal > value);
                bin |= split << level;
            }
            const ui32 writeIdx = writeIndices ? writeIndices[tid] : tid;
            cursor[writeIdx] = bin;
            tid += blockDim.x  * gridDim.x;
        }
    }


    //doc parallel routines
    void AddObliviousTree(const TCFeature* features, const ui8* bins, const float* leaves, ui32 depth,
                          const ui32* cindex,
                          const ui32* readIndices,
                          const ui32* writeIndices,
                          float* cursor,
                          ui32 size,
                          TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide<ui32>(size, blockSize);
        AddObliviousTreeImpl<< <numBlocks, blockSize, 0, stream>>>(features, bins, leaves, depth, cindex, readIndices, writeIndices, cursor, size);
    }


    void ComputeObliviousTreeBins(const TCFeature* features, const ui8* bins, ui32 depth,
                                  const ui32* cindex,
                                  const ui32* readIndices,
                                  const ui32* writeIndices,
                                  ui32* cursor,
                                  ui32 size,
                                  TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide<ui32>(size, blockSize);
       ComputeObliviousTreeBinsImpl<< <numBlocks, blockSize, 0, stream>>>(features, bins, depth, cindex, readIndices, writeIndices, cursor, size);
    }

}
