#include "hip/hip_runtime.h"
#include "ctr_calcers.cuh"
#include <catboost/cuda/cuda_util/kernel/index_wrapper.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>


namespace NKernel {

    __global__ void GatherTrivialWeightsImpl(const ui32* indices, ui32 size,
                                             ui32 firstZeroIndex, bool writeSegmentStartFloatMask,
                                             float* dst) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size) {
            TIndexWrapper indexWrapper(StreamLoad(indices + i));
            const float val = (indexWrapper.Index() < firstZeroIndex ? 1.0f : 0.0f);
            dst[i] = (writeSegmentStartFloatMask && indexWrapper.IsSegmentStart()) ? -val : val;
        }
    }

    void GatherTrivialWeights(const ui32* indices, ui32 size,
                              ui32 firstZeroIndex, bool writeSegmentStartFloatMask,
                              float* dst,
                              TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            GatherTrivialWeightsImpl <<<numBlocks, blockSize, 0, stream>>>(indices, size, firstZeroIndex, writeSegmentStartFloatMask, dst);
        }
    }


    __global__ void WriteMaskImpl(const ui32* indices, ui32 size,
                                  float* dst) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size) {
            TIndexWrapper indexWrapper(StreamLoad(indices + i));
            const float val = dst[i];
            dst[i] = indexWrapper.IsSegmentStart() ? -val : val;
        }
    }

    void WriteMask(const ui32* indices, ui32 size,
                   float* dst,
                   TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            WriteMaskImpl <<<numBlocks, blockSize, 0, stream>>>(indices, size, dst);
        }
    }

    __global__ void WeightedBinFreqCtrsImpl(const ui32* writeIndices, const ui32* bins,
                                            const float* binSums,
                                            float totalWeight, float prior, float priorObservations,
                                            float* dst,
                                            ui32 size) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size) {
            ui32 dstIdx = writeIndices ? TIndexWrapper(writeIndices[i]).Index() : i;
            dst[dstIdx] = (binSums[bins[i]] + prior) / (totalWeight + priorObservations);
        }
    }

    template <int BLOCK_SIZE, int DOCS_PER_THREAD>
    __global__ void NonWeightedBinFreqCtrsImpl(const ui32* writeIndices,
                                               const ui32* bins, const ui32* binOffsets, ui32 size,
                                               float prior, float priorObservations,
                                               float* dst) {
        const ui32 i = blockIdx.x * BLOCK_SIZE * DOCS_PER_THREAD  + threadIdx.x;

        int dstIndices[DOCS_PER_THREAD];
        ui32 binsLocal[DOCS_PER_THREAD];

        #pragma unroll DOCS_PER_THREAD
        for (int j = 0; j < DOCS_PER_THREAD; ++j) {
            const ui32 idx = i + BLOCK_SIZE * j;
            dstIndices[j] = idx < size ? (int)(writeIndices ? TIndexWrapper(writeIndices[idx]).Index() : idx) : -1;
            binsLocal[j] =  idx < size ? bins[idx] : 0;
        }

        #pragma unroll DOCS_PER_THREAD
        for (int j = 0; j < DOCS_PER_THREAD; ++j)
        {
            const ui32 bin = binsLocal[j];
            const ui32 currentBinOffset = LdgWithFallback(binOffsets + bin, 0);
            const ui32 nextBinOffset = bin < size ? LdgWithFallback(binOffsets + bin + 1, 0) : size;
            binsLocal[j] = (nextBinOffset - currentBinOffset);
        }

        #pragma unroll DOCS_PER_THREAD
        for (int j = 0; j < DOCS_PER_THREAD; ++j)
        {
            if (dstIndices[j] != -1)
            {
                WriteThrough(dst + dstIndices[j], (binsLocal[j] + prior) / (size + priorObservations));
            }
        }
    }


    void ComputeWeightedBinFreqCtr(const ui32* writeIdx, const ui32* bins,
                                   const float* binSums,
                                   float totalWeight, float prior, float priorObservations,
                                   float* dst,
                                   ui32 size, TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            WeightedBinFreqCtrsImpl <<<numBlocks, blockSize, 0, stream>>>(writeIdx, bins, binSums, totalWeight, prior, priorObservations, dst, size);
        }
    }

    void ComputeNonWeightedBinFreqCtr(const ui32* writeIdx, const ui32* bins,
                                      const ui32* binOffsets, ui32 size,
                                      float prior, float priorObservations,
                                      float* dst, TCudaStream stream) {
        const ui32 blockSize = 256;
        const ui32 elementsPerThreads = 4;
        const ui32 numBlocks = CeilDivide(size, blockSize * elementsPerThreads);
        if (numBlocks) {
            NonWeightedBinFreqCtrsImpl<blockSize, elementsPerThreads> <<<numBlocks, blockSize, 0, stream>>>(writeIdx, bins, binOffsets, size, prior, priorObservations, dst);
        }
    }


    __global__ void UpdateBordersMaskImpl(const ui32* bins, const ui32* prevBins, ui32* indices, ui32 size) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size) {
            TIndexWrapper currentIndex(indices[i]);

            bool mask = currentIndex.IsSegmentStart();
            if (!mask) {
                mask |= (i == 0 || (bins[i] != bins[i - 1]));
            }
            if (!mask) {
                TIndexWrapper prevIndex(indices[i - 1]);
                const ui32 currentBin = LdgWithFallback(prevBins, currentIndex.Index());
                const ui32 prevBin = LdgWithFallback(prevBins, prevIndex.Index());
                mask |=  currentBin != prevBin;
            }
            currentIndex.UpdateMask(mask);
            indices[i] = currentIndex.Value();
        }
    }

    void UpdateBordersMask(const ui32* bins, const ui32* prevBins, ui32* indices, ui32 size, TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);

        if (numBlocks) {
            UpdateBordersMaskImpl<<<numBlocks, blockSize, 0, stream>>>(bins, prevBins, indices, size);
        }
    }


    __global__ void MergeBinsKernelImpl(ui32* bins, const ui32* prev, ui32 shift, ui32 size) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size) {
            bins[i] = (bins[i] << shift) | prev[i];
        }
    }

    void MergeBinsKernel(ui32* bins, const ui32* prev, ui32 shift, ui32 size, TCudaStream stream)  {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);

        if (numBlocks) {
            MergeBinsKernelImpl<<<numBlocks, blockSize, 0, stream>>>(bins, prev, shift, size);
        }
    }


    __global__ void ExtractBorderMasksStartImpl(const ui32* indices, ui32* dst, ui32 size) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size) {
            const bool isStart = TIndexWrapper(indices[i]).IsSegmentStart();
            dst[i] = isStart;
        }
    }

    __global__ void ExtractBorderMasksEndImpl(const ui32* indices, ui32* dst, ui32 size) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < size) {
            const bool isEnd= (i + 1) < size ? TIndexWrapper(indices[i + 1]).IsSegmentStart() : true;
            dst[i] = isEnd;
        }
    }

    void ExtractBorderMasks(const ui32* indices, ui32* dst, ui32 size, bool startSegment, TCudaStream stream) {
        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            if (startSegment) {
                ExtractBorderMasksStartImpl << < numBlocks, blockSize, 0, stream >> > (indices, dst, size);
            } else {
                ExtractBorderMasksEndImpl << < numBlocks, blockSize, 0, stream >> > (indices, dst, size);
            }
        }
    }


    template <bool IS_BORDERS, int N>
    __global__ void FillBinarizedTargetsStatsImpl(const ui8* binarizedTargets, const float* sampleWeight,
                                                  float* dst, ui32 size, ui32 binIndex) {

        const ui32 i = (blockIdx.x * blockDim.x + threadIdx.x) * N;

        float localSamples[N];

#pragma unroll
        for (int k = 0; k < N; ++k) {
            const int idx = i + k;

            if (idx < size) {
                const float weight = StreamLoad(sampleWeight + idx);
                localSamples[k]=  abs(weight) * (IS_BORDERS ? StreamLoad(binarizedTargets + idx) > binIndex
                                                            : StreamLoad(binarizedTargets + idx) == binIndex);
                localSamples[k] = ExtractSignBit(weight) ? -localSamples[k] : localSamples[k];
            }
        }

#pragma unroll
        for (int k = 0; k < N; ++k) {
            const int idx = i + k;
            if (idx < size) {
                dst[idx] = localSamples[k];
            }
        }
    }

    void FillBinarizedTargetsStats(const ui8* sample, const float* sampleWeights, ui32 size,
                                   float* sums, ui32 binIndex, bool borders,
                                   TCudaStream stream) {
        const ui32 blockSize = 256;
        const int N = 4;
        const ui32 numBlocks = CeilDivide(size, N * blockSize);
        if (numBlocks) {
            if (borders) {
                FillBinarizedTargetsStatsImpl<true, N> << < numBlocks, blockSize, 0, stream >> >(sample, sampleWeights, sums, size, binIndex);
            } else {
                FillBinarizedTargetsStatsImpl<false, N> << < numBlocks, blockSize, 0, stream >> >(sample, sampleWeights, sums, size, binIndex);
            }
        }
    }


    __global__ void MakeMeansImpl(float* sums, const float* weights, ui32 size,
                                  float sumPrior, float weightPrior) {
        ui32 tid = blockDim.x * blockIdx.x + threadIdx.x;

        if (tid < size) {
            sums[tid] = (sums[tid] + sumPrior) / (weights[tid] + weightPrior);
        }
    }

    void MakeMeans(float* sums, const float* weights, ui32 size,
                   float sumPrior, float weightPrior,
                   TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            MakeMeansImpl <<<numBlocks, blockSize, 0, stream>>> (sums, weights, size, sumPrior, weightPrior);
        }
    }

    __global__ void MakeMeansAndScatterImpl(const float* sums, const float* weights, ui32 size,
                                            float sumPrior, float weightPrior,
                                            const ui32* map, ui32 mask,
                                            float* dst) {
        const ui32 i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size) {
            const ui32 m = map ? StreamLoad(map + i) & mask : i;
            dst[m] = (sums[i] + sumPrior) / (weights[i] + weightPrior);
        }
    }

    void MakeMeansAndScatter(const float* sums, const float* weights, ui32 size,
                             float sumPrior, float weightPrior,
                             const ui32* map, ui32 mask,
                             float* dst,
                             TCudaStream stream) {

        const ui32 blockSize = 256;
        const ui32 numBlocks = CeilDivide(size, blockSize);
        if (numBlocks) {
            MakeMeansAndScatterImpl <<<numBlocks, blockSize, 0, stream>>> (sums, weights, size, sumPrior, weightPrior, map, mask, dst);
        }
    }
}
