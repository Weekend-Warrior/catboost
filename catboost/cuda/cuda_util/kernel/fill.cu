#include "hip/hip_runtime.h"
#include "fill.cuh"
#include <catboost/cuda/cuda_lib/kernel/arch.cuh>

namespace NKernel
{

    template<typename T>
    __global__ void FillBufferImpl(T* buffer, T value, ui64  size)
    {
        ui64 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size)
        {
            buffer[i] = value;
            i += gridDim.x * blockDim.x;
        }
    }

    template<typename T>
    void FillBuffer(T* buffer, T value, ui64 size, TCudaStream stream)
    {
        if (size > 0)
        {
            const ui32 blockSize = 512;
            const ui64 numBlocks = min((size + blockSize - 1) / blockSize,
                                         (ui64)TArchProps::MaxBlockCount());
            FillBufferImpl<T> << < numBlocks, blockSize, 0, stream>> > (buffer, value, size);
        }
    }

    template<typename T>
    __global__ void MakeSequenceImpl(T offset, T* buffer, ui64  size)
    {
        ui64 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size) {
            buffer[i] = offset + i;
            i += gridDim.x * blockDim.x;
        }
    }

    template<typename T>
    void MakeSequence(T offset, T* buffer, ui64  size, TCudaStream stream)
    {
        if (size > 0)
        {
            const ui32 blockSize = 512;
            const ui64 numBlocks = min((size + blockSize - 1) / blockSize,
                                         (ui64)TArchProps::MaxBlockCount());
            MakeSequenceImpl<T> << < numBlocks, blockSize, 0, stream >> > (offset, buffer, size);
        }
    }

    template<typename T>
    __global__ void InversePermutationImpl(const T* indices, T* dst, ui64 size) {
        ui64 i = blockIdx.x * blockDim.x + threadIdx.x;
        while (i < size) {
            dst[indices[i]] = i;
            i += gridDim.x * blockDim.x;
        }
    }

    template<typename T>
    void InversePermutation(const T* order, T* inverseOrder, ui64 size, TCudaStream stream)
    {
        if (size > 0)
        {
            const ui32 blockSize = 512;
            const ui64 numBlocks = min((size + blockSize - 1) / blockSize,
                                       (ui64)TArchProps::MaxBlockCount());
            InversePermutationImpl<T> << < numBlocks, blockSize, 0, stream >> > (order, inverseOrder, size);
        }
    }



    template void FillBuffer<char>(char* buffer, char value, ui64  size, TCudaStream stream);

    template void FillBuffer<unsigned char>(unsigned char* buffer, unsigned char value, ui64  size, TCudaStream stream);

    template void FillBuffer<short>(short* buffer, short value, ui64  size, TCudaStream stream);

    template void FillBuffer<ui16>(ui16* buffer, ui16 value, ui64  size, TCudaStream stream);

    template void FillBuffer<int>(int* buffer, int value, ui64  size, TCudaStream stream);

    template void FillBuffer<ui32>(ui32* buffer, ui32 value, ui64  size, TCudaStream stream);

    template void FillBuffer<float>(float* buffer, float value, ui64  size, TCudaStream stream);

    template void FillBuffer<double>(double* buffer, double value, ui64  size, TCudaStream stream);

    template void FillBuffer<long>(long* buffer, long value, ui64  size, TCudaStream stream);

    template void FillBuffer<ui64>(ui64* buffer, ui64 value, ui64  size, TCudaStream stream);

    template void MakeSequence<int>(int offset, int* buffer, ui64  size, TCudaStream stream);
    template void MakeSequence<ui32>(ui32 offset, ui32* buffer, ui64  size, TCudaStream stream);
    template void MakeSequence<ui64>(ui64 offset, ui64* buffer, ui64  size, TCudaStream stream);

    template void InversePermutation<ui32>(const ui32* order, ui32* inverseOrder, ui64 size, TCudaStream stream);
    template void InversePermutation<int>(const int* order, int* inverseOrder, ui64 size, TCudaStream stream);
}
