#include "hip/hip_runtime.h"
#include "filter.cuh"

namespace NKernel {

    struct TZeroWeightFilter {

        __device__ ui32 operator()(float w) {
            return abs(w) > 1e-20f;
        }
    };

    template <class Filter = TZeroWeightFilter>
    __global__ void FilterImpl(const float* weights,
                               int size,
                               ui32* result) {
        const int i = blockIdx.x * blockDim.x + threadIdx.x;
        Filter filter;
        if (i < size) {
            result[i] = filter(weights[i]);
        }
    }


    void Filter(const float* weights, const ui32 size, ui32* result, TCudaStream stream) {
        if (size > 0) {
            const ui32 blockSize = 512;
            const ui32 numBlocks = (size + blockSize - 1) / (blockSize);
            FilterImpl << <numBlocks, blockSize, 0, stream>>>(weights, size, result);
        }
    }
}
