#include "segmented_sort.cuh"
#include <contrib/libs/cub/cub/device/device_segmented_radix_sort.cuh>

namespace NKernel {

    template <typename K, typename V>
    hipError_t SegmentedRadixSort(K* keys, V* values,
                                   K* tmpKeys, V* tmpValues,
                                   int size,
                                   const ui32* segmentStarts, const ui32* segmentEnds,
                                   int numSegments,
                                   TSegmentedRadixSortContext& context, TCudaStream stream)
    {
        hipcub::DoubleBuffer<K> doubleBufferKeys(keys, tmpKeys);
        hipError_t error;
        int* starts = const_cast<int*>((const int*)(segmentStarts));
        int* ends = const_cast<int*>((const int*)(segmentEnds));

        if (values) {
            hipcub::DoubleBuffer<V> doubleBufferValues(values, tmpValues);

            if (context.Descending) {


                hipcub::DoubleBuffer<K> inputValues;
                error = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(context.TempStorage, context.TempStorageSize,
                                                                           doubleBufferKeys,
                                                                           doubleBufferValues,
                                                                           size,
                                                                           numSegments,
                                                                           starts, ends,
                                                                           context.FirstBit, context.LastBit,
                                                                           stream);
            } else {
                error = hipcub::DeviceSegmentedRadixSort::SortPairs(context.TempStorage, context.TempStorageSize,
                                                                 doubleBufferKeys,
                                                                 doubleBufferValues,
                                                                 size,
                                                                 numSegments,
                                                                 starts, ends,
                                                                 context.FirstBit, context.LastBit,
                                                                 stream);
            }

            if (doubleBufferValues.Current() != values) {
                hipMemcpyAsync(values, doubleBufferValues.Current(), sizeof(V) * size, hipMemcpyDefault, stream);
            }
        } else {
            if (context.Descending) {
                error = hipcub::DeviceSegmentedRadixSort::SortKeysDescending(context.TempStorage, context.TempStorageSize,
                                                                          doubleBufferKeys,
                                                                          size,
                                                                          numSegments,
                                                                          starts, ends,
                                                                          context.FirstBit, context.LastBit,
                                                                          stream);
                } else {
                    error = hipcub::DeviceSegmentedRadixSort::SortKeys(context.TempStorage, context.TempStorageSize,
                                                                    doubleBufferKeys,
                                                                    size,
                                                                    numSegments,
                                                                    starts, ends,
                                                                    context.FirstBit, context.LastBit,
                                                                    stream);
                }
        }
        //TODO(noxoomo): error handling
        if (doubleBufferKeys.Current() != keys) {
            hipMemcpyAsync(keys, doubleBufferKeys.Current(), sizeof(K) * size, hipMemcpyDefault, stream);
        }
        return error;
    }



    template hipError_t SegmentedRadixSort(ui32* keys, ui32* values,
                                            ui32* tmpKeys, ui32* tmpValues, int size,
                                            const ui32* segmentStarts, const ui32* segmentEnds, int segmentsCount,
                                            TSegmentedRadixSortContext& context, TCudaStream stream);




}
